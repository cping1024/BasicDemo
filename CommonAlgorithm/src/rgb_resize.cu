#include "hip/hip_runtime.h"

/// @brief 数字图像双线性插值

#include <stdio.h>

#include <iostream>
#include <string>
#include <chrono>

#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/gpu/gpu.hpp>

using namespace std::chrono;

typedef struct rect {
    int x;
    int y;
    int w;
    int h;
} sn_rect;

typedef struct size {
    int width;
    int height;
} sn_size;

typedef struct color{
    unsigned char r;
    unsigned char g;
    unsigned char b;
    unsigned char a;
} sn_color;

int divUp(int N, int M) { return (N - 1) / M + 1;}

__global__ void rgb_resize_kenerl(const unsigned char* in, unsigned char* out, int i_w, int i_h, int o_w, int o_h, float x_scale, float y_scale) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z;

    if (idx >= o_w || idy >= o_h) {
        return;
    }

    float fx = (idx + 0.5) * i_w / o_w - 0.5;
    float fy = (idy + 0.5) * i_h / o_h - 0.5;

    int px1 = int(fx);
    int py1 = int(fy);
    int px2 = px1 + 1;
    if (px2 >= iw) {
        --px2;
    }
    int py2 = py1;
    int px3 = px1;
    int py3 = py1 + 1;
    if (py3 >= ih) {
        --py3;
    }
    int px4 = px2;
    int py4 = py3;

    float pv1 = fabs(px2 - fx) * fabs(py3 - fy);
    float pv2 = fabs(px1 - fx) * fabs(py3 - fy);
    float pv3 = fabs(px2 - fx) * fabs(fy - py1);
    float pv4 = fabs(px1 - fx) * fabs(fy - py1);
    int offset = (idy * o_w + idx) * 3 + idz;
    int offset1 = (py1 * i_w + px1) * 3 + idz;
    int offset2 = (py2 * i_w + px2) * 3 + idz;
    int offset3 = (py3 * i_w + px3) * 3 + idz;
    int offset4 = (py4 * i_w + px4) * 3 + idz;
    
    out[offset] = pv1 * in[offset1] + pv2 * in[offset2] + pv3 * in[offset3] + pv4 * in[offset4];
}

void sn_gpu_resize(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& des, const cv::Size& size) {

    if (!src.data) {
        return;
    }

    void *d_src, *d_des;
    int channel = src.channels();
    int owidth = size.width;
    int oheight = size.height;
    hipMalloc(&d_src, src.cols * src.rows * channel);
    hipMalloc(&d_des, owidth * oheight * channel);
    hipMemcpy2D(d_src, src.cols * channel, src.data, src.step, src.cols * channel, src.rows, hipMemcpyDefault);
    
    const int N = 16;
    dim3 block(N, N, 1);
    dim3 grid(divUp(owidth, N), divUp(oheight, N), 3);
    float scalex = float(src.cols) / float(size.width);
    float scaley = float(src.rows) / float(size.height);
    rgb_resize_kenerl<<<grid, block>>>((const unsigned char*)d_src, (unsigned char*)d_des, src.cols, src.rows, owidth, oheight, scalex, scaley);
    hipDeviceSynchronize();
    
    cv::Size osize(owidth, oheight);
    if (des.size() != osize) {
        des.create(osize, src.type());
    }

    hipMemcpy2D(des.data, des.step, d_des, owidth * channel, owidth * channel, oheight, hipMemcpyDefault);

    hipFree(d_des);
    hipFree(d_src);
}

__global__ void copyMakeBorder_kernel(const unsigned char* in, unsigned char* out,
                                        int iw, int ih,
                                        int ow, int oh,
                                        int left, int top, int right, int bottom,
                                        unsigned char b, unsigned char g, unsigned char r )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= ow || idy >= oh) {
        return;
    }

    int offsetx = idx - left;
    int offsety = idy - top;
    int offset = (idy * ow + idx) * 3;
    int offset1 = (offsety * iw + offsetx) * 3;

    if ((0 <= offsetx) && (offsetx <= iw) && (0 <= offsety) && (offsety <= ih)) {
        out[offset] = in[offset1];
        out[offset + 1] = in[offset1 + 1];
        out[offset + 2] = in[offset1 + 2];
    } else {

        out[offset] = b;
        out[offset + 1] = g;
        out[offset + 2] = r;
    }
}

void copyMakeborder_cpu(const unsigned char* in, unsigned char* out, sn_size* isize, int left, int top, int right, int bottom, sn_size* osize, sn_color* color) {

    if (!in || !out) {
        return;
    }

    for (int i = 0; i < osize->height; ++i) {
        for (int j = 0; j < osize->width; ++j) {
           
           int offsetx = j - left;
           int offsety = i - top;
           if ((0 <= offsetx) && (offsetx <= isize->width) && (0 <= offsety) && (offsety <= isize->height)) {
                out[(i * osize->width + j) * 3] = in[(offsety * isize->width + offsetx) * 3];
                out[(i * osize->width + j) * 3 + 1] = in[(offsety * isize->width + offsetx) * 3 + 1];
                out[(i * osize->width + j) * 3 + 2] = in[(offsety * isize->width + offsetx) * 3 + 2];
           } else { 
                out[(i * osize->width + j) * 3] = color->b;
                out[(i * osize->width + j) * 3 + 1] = color->g;
                out[(i * osize->width + j) * 3 + 2] = color->r;
           }
        }
    }
}

void rgb_resize_cpu(const unsigned char* src, unsigned char* des, int iw, int ih, int ow, int oh, float x, float y)
{
    if (!src || !des) {
        return; 
    }
    
    for (int i = 0; i < oh; ++i) {        
        float fy = (i + 0.5) * ih / oh - 0.5;
        for (int j = 0; j < ow; ++j) {
            float fx = (j + 0.5) * iw / ow - 0.5;

            int px1 = int(fx);
            int py1 = int(fy);
            int px2 = px1 + 1;
            if (px2 >= iw) {
                --px2;
            }

            int py2 = py1;
            int px3 = px1;
            int py3 = py1 + 1;
            if (py3 >= ih) {
                --py3;
            }
            int px4 = px2;
            int py4 = py3;

            float pv1 = fabs(px2 - fx) * fabs(py3 - fy);
            float pv2 = fabs(px1 - fx) * fabs(py3 - fy);
            float pv3 = fabs(px2 - fx) * fabs(fy - py1);
            float pv4 = fabs(px1 - fx) * fabs(fy - py1);

            for (int n = 0; n < 3; ++n) {
                int offset1 = py1 * iw * 3 + px1 * 3 + n;
                int offset2 = py2 * iw * 3 + px2 * 3 + n;
                int offset3 = py3 * iw * 3 + px3 * 3 + n;
                int offset4 = py4 * iw * 3 + px4 * 3 + n;
                des[(i * ow  + j) * 3 + n] = pv1 * src[offset1] + pv2 * src[offset2] + pv3 * src[offset3] + pv4 * src[offset4];
            }
        }
    }

}

void sn_cpu_resize(const cv::Mat& src, cv::Mat& des, const cv::Size& size){
    if (!src.data) {
        return;
    }

    float scalex = float(src.cols) / float(size.width);
    float scaley = float(src.rows) / float(size.height);
    if (des.size() != size) {
        des.create(size, src.type());
    }

    rgb_resize_cpu(src.data, des.data, src.cols, src.rows, size.width, size.height, scalex, scaley);    
}

void test_resize_kernel(std::string& filename){
    cv::Mat image = cv::imread(filename);
    if (!image.data) {
        return;
    }

    const int N = 16;
    const int des_w = 320;
    const int des_h = 180;
    const float x_scale = float(image.cols) / float(des_w);
    const float y_scale = float(image.rows) / float(des_h);

    void* src;
    const int len = image.cols * image.rows * image.channels();
    hipMalloc(&src, len);
    hipMemcpy(src, image.data, len, hipMemcpyHostToDevice);

    void* rgb;
    hipMalloc(&rgb, des_w * des_h * 3);

steady_clock::time_point start = steady_clock::now();
    dim3 block(N, N, 1);
    dim3 grid(divUp(des_w, N), divUp(des_h, N), 3);
    rgb_resize_kenerl<<<grid, block>>>((const unsigned char*)src, (unsigned char*)rgb, image.cols, image.rows, des_w, des_h, x_scale, y_scale);
    hipDeviceSynchronize();
steady_clock::time_point stop = steady_clock::now();
milliseconds time = duration_cast<milliseconds>(stop - start);
printf("gpu resize time[%ld]ms.\n", time.count());

    cv::Mat resize_image;
    resize_image.create(cv::Size(des_w, des_h), image.type());
    hipMemcpy(resize_image.data, rgb, des_w * des_h * 3, hipMemcpyDeviceToHost);

    cv::imshow("src", image);
    cv::waitKey();

    cv::imshow("resize", resize_image);
    cv::waitKey();

    cv::Mat cpu_resize_image;
    cpu_resize_image.create(cv::Size(des_w, des_h), image.type());

start = steady_clock::now();
    rgb_resize_cpu(image.data, cpu_resize_image.data, image.cols, image.rows, des_w, des_h, x_scale, y_scale);
stop = steady_clock::now();
time = duration_cast<milliseconds>(stop - start);
printf("cpu resize time[%ld]ms.\n", time.count());    

    cv::imshow("cpu", cpu_resize_image);
    cv::waitKey();

    hipFree(rgb);
    hipFree(src);
}

void test_copymakeborder(std::string& filename){
    cv::Mat image = cv::imread(filename);
    if (!image.data) {
        printf("read image fail.\n");
        return;
    }

    const int N = 16;
    const int left = 10;
    const int top = 10;
    const int right = 10;
    const int bottom = 10;
    
    int owidth = image.cols + left + right;
    int oheight = image.rows + top + bottom;

    void *src;
    hipMalloc(&src, image.cols * image.rows * image.channels());
    hipMemcpy(src, image.data, image.cols * image.rows * image.channels(), hipMemcpyHostToDevice);

    void *des;
    hipMalloc(&des, owidth * oheight * image.channels());

    dim3 block(N, N);
    dim3 grid(divUp(owidth, N), divUp(oheight, N));
    copyMakeBorder_kernel<<<grid, block>>>((const unsigned char*)src, (unsigned char*)des, image.cols, image.rows, owidth, oheight, left, top, right, bottom, 0, 0, 255);
    hipDeviceSynchronize();

    cv::Mat border_image;
    border_image.create(cv::Size(owidth, oheight), image.type());
    //copyMakeborder_cpu((const unsigned char*)image.data, (unsigned char*)border_image.data, &isize, left, top, right, bottom, &osize, &color);
    hipMemcpy(border_image.data, des, owidth * oheight * 3, hipMemcpyDeviceToHost);
    cv::imshow("border", border_image);
    cv::waitKey();

    cv::imshow("src", image);
    cv::waitKey();

    hipFree(src);
    hipFree(des);
}

void test_sn_gpu_resize(const std::string& filename, float scale, int row) {
    
    cv::Mat image = cv::imread(filename);
    if (!image.data) {
        return;
    }

    cv::gpu::GpuMat gpu_mat(image);
    
    printf("gpu mat width[%d], height[%d], step[%d].\n", gpu_mat.cols, gpu_mat.rows, gpu_mat.step);
    const int count = 1;    
    //float scale = 0.33f;
    int width = image.cols * scale;
    int height = image.rows * scale;

    steady_clock::time_point start = steady_clock::now();
    for (int i = 0; i < count; ++i) {
        cv::gpu::GpuMat resize_image;
        sn_gpu_resize(gpu_mat, resize_image, cv::Size(width, height));    
    }
    steady_clock::time_point stop = steady_clock::now();
    milliseconds time = duration_cast<milliseconds>(stop - start);
    printf("sn gpu resize avg time[%f]ms.\n", (time.count() * 1.0f) / count);

    cv::gpu::GpuMat resize_image;
    sn_gpu_resize(gpu_mat, resize_image, cv::Size(width, height));

    cv::Mat temp;
    //sn_cpu_resize(image, temp, cv::Size(width, height));
    resize_image.download(temp);
    cv::imshow("temp", temp);
    cv::waitKey();
    std::cout << "gpu resize image:" << temp.row(row) << std::endl;
    printf("resize image w[%d], h[%d], scale[%f], show row[%d].\n", temp.cols, temp.rows, scale, row);

    cv::imshow("src", image);
    cv::waitKey();
}

int main(int argc, char* argv[]) {

    if (argc != 4) {
        printf("usage:./application <image filename> scale row_num.\n");
        return -1;    
    }

    std::string filename(argv[1]);	
    //test_resize_kernel(filename);

    //test_copymakeborder(filename);

    float scale = atof(argv[2]);
    int row = atof(argv[3]);
    test_sn_gpu_resize(argv[1], scale, row);
    return 0;
}

